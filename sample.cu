#include "hip/hip_runtime.h"

__device__ half dev_square(half in){
	float4 f4 = make_float4(0.0f, 1.0f, 2.0f, 3.0f);
	return in * in;
}

template<int N>
__global__ void kernel(half* out_ptr, const half* in_ptr){
	int tid = blockIdx.x + blockDim.x + threadIdx.x;
	if(tid >= N)return;
	out_ptr[tid] = dev_square(in_ptr[tid]);
}
