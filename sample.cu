#include "hip/hip_runtime.h"

__device__ half dev_square(half in){
	return in * in;
}

template<int N>
__global__ void kernel(half* out_ptr, const half* in_ptr){
	int tid = blockIdx.x + blockDim.x + threadIdx.x;
	if(tid >= N)return;
	out_ptr[tid] = dev_square(in_ptr[tid]);
}
